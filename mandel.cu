#include "hip/hip_runtime.h"
#include "mandel.cuh"

__device__ int get_inter (thrust::complex<float> c) {
    int i;
    thrust::complex<float> z(0.0, 0.0);

    for (i = 0; i< INTER_LIMIT; ++i) {
        if (thrust::norm(z) > 4 ) {
            break;
        }
        z = thrust::pow(z, 2) + c;
    }
    return i;
}

__global__ void fill_matrix (int *res, const int w, const int h, thrust::complex<float> c0, const float del_y, const float del_x, const int threads, const int blocks, const int offset) {
    thrust::complex<float> del(0, 0);
    int k = threadIdx.x + blockIdx.x*threads + blocks*threads*offset;
    if (k >= w*h)
        return;
    del.real(del_x * (k%w));
    del.imag(del_y * (k/w));
    res[k] = get_inter(c0 + del);
    return;
}

__host__ void prepare (int *res_matrix, const int w, const int h, thrust::complex<float> c0, const float del_y, const float del_x, const int threads) {
    int *d_res_matrix; 
    int *d_w; 
    int *d_h;
    thrust::complex<float> *d_c0; 
    float *d_del_y; 
    float *d_del_x; 
    
    hipSetDevice(0);

    hipMallocManaged((void **) &d_res_matrix, sizeof(int)*w*h);
    hipMallocManaged((void **) &d_w, sizeof(int));
    hipMallocManaged((void **) &d_h, sizeof(int));
    hipMallocManaged((void **) &d_c0, sizeof(thrust::complex<float>));
    hipMallocManaged((void **) &d_del_y, sizeof(float));
    hipMallocManaged((void **) &d_del_x, sizeof(float));
    
    hipMemcpy(d_w, &w, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_h, &h, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_c0, &c0, sizeof(thrust::complex<float>), hipMemcpyHostToDevice);
    hipMemcpy(d_del_y, &del_y, sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_del_x, &del_x, sizeof(float), hipMemcpyHostToDevice);
    
    int block = 1024;
    int max = ((w*h) / (threads*block)) + 1;
    for (int i = 0; i < max; ++i) {
        fill_matrix<<<block, threads>>> (d_res_matrix, *d_w, *d_h, *d_c0, *d_del_y, *d_del_x, threads, block, i);
        hipDeviceSynchronize();
    }
    
    hipMemcpy(res_matrix, d_res_matrix, sizeof(int)*w*h, hipMemcpyDeviceToHost);
    
    hipFree(d_res_matrix);
    hipFree(d_w);
    hipFree(d_h);
    hipFree(d_c0);
    hipFree(d_del_y);
    hipFree(d_del_x);
    return;
}
