#include "hip/hip_runtime.h"
#include "mandel.cuh"

__device__ int get_inter (thrust::complex<float> c) {
    int i;
    thrust::complex<float> z(0.0, 0.0);

    for (i = 0; i< INTER_LIMIT; ++i) {
        if (thrust::norm(z) > 4 ) {
            break;
        }
        z = thrust::pow(z, 2) + c;
    }
    return i;
}

__global__ void fill_matrix (int *res_matrix, const int w, const int h, thrust::complex<float> c0, const float del_y, const float del_x) {
    thrust::complex<float> del(0, 0);
    for (int i = 0; i < h; i += 1) {
        for (int j = 0; j < w; j += 1) {
            del.real(del_x * j);
            del.imag(del_y * i);
            res_matrix[i*w + j] = get_inter(c0 + del);
        }
    }
    return;
}

__host__ void prepare (int *res_matrix, const int w, const int h, thrust::complex<float> c0, const float del_y, const float del_x, const int threads) {
    int *d_res_matrix; 
    int *d_w; 
    int *d_h;
    thrust::complex<float> *c_c0; 
    float *d_del_y; 
    float *d_del_x; 
    hipMalloc((void **) &d_res_matrix, sizeof(int)*w*h);
    hipMalloc((void **) &d_w, sizeof(int));
    hipMalloc((void **) &d_h, sizeof(int));
    hipMalloc((void **) &c_c0, sizeof(thrust::complex<float>));
    hipMalloc((void **) &d_del_y, sizeof(float));
    hipMalloc((void **) &d_del_x, sizeof(float));
    hipMemcpy(d_w, &w, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_h, &h, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(c_c0, &c0, sizeof(thrust::complex<float>), hipMemcpyHostToDevice);
    hipMemcpy(d_del_y, &del_y, sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_del_x, &del_x, sizeof(float), hipMemcpyHostToDevice);
    //fill_matrix<<<1, 1>>> (d_res_matrix, d_w, d_h, d_c0, d_del_y, d_del_x);
    return;
}